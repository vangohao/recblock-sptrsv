#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "common.h"
#include "mmio.h"
#include "mmio_highlevel.h"
#include "recblocking_solver.h"
#include "recblocking_solver_cuda.h"

#include "unisolver/ArrayUtils.hpp"
#include "unisolver/JsonUtils.hpp"

using namespace uni;

#define CHECK_CUDA(func)                                               \
    {                                                                  \
        hipError_t status = (func);                                   \
        if (status != hipSuccess) {                                   \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipGetErrorString(status), status);      \
            while (1);                                                 \
        }                                                              \
    }

#define CHECK_CUSPARSE(func)                                               \
    {                                                                      \
        hipsparseStatus_t status = (func);                                  \
        if (status != HIPSPARSE_STATUS_SUCCESS) {                           \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipsparseGetErrorString(status), status);      \
            while (1);                                                     \
        }                                                                  \
    }

using cusp_int = int;
#define my_CUSPARSE_INDEX HIPSPARSE_INDEX_32I

#define MAX_DOF_TEST 8

struct benchmark_record {
    double total_time = 0;
    long flops = 0;
    long bytes = 0;
    long count = 0;
};

benchmark_record benchmark_record_map_lower[MAX_DOF_TEST];

double recblock_sptrsv_csr(int m, int n, int nnzTR, int *csrRowPtrTR,
                           int *csrColIdxTR, VALUE_TYPE *csrValTR,
                           VALUE_TYPE *b, VALUE_TYPE *x, int lv) {
    int device_id = 0;
    if (lv == -1) {
        int li = 1;
        for (li = 1; li <= 100; li++) {
            if (m / pow(2, (li + 1)) <
                (device_id == 0 ? 92160
                                : 58880))  // 92160 (4608x20) is titan rtx,
                                           // 58880 (2944x20) is rtx 2080
                break;
        }
        lv = li;
    }

    int rhs = 1;
    int substitution = SUBSTITUTION_FORWARD;

    // transpose CSR of U and L to CSC
    int *cscColPtrTR = (int *)malloc(sizeof(int) * (n + 1));
    cscColPtrTR[0] = 0;
    int *cscRowIdxTR = (int *)malloc(sizeof(int) * nnzTR);
    VALUE_TYPE *cscValTR = (VALUE_TYPE *)malloc(sizeof(VALUE_TYPE) * nnzTR);
    matrix_transposition(m, n, nnzTR, csrRowPtrTR, csrColIdxTR, csrValTR,
                         cscRowIdxTR, cscColPtrTR, cscValTR);

    if (lv == -1) {
        int li = 1;
        for (li = 1; li <= 100; li++) {
            if (m / pow(2, (li + 1)) <
                (device_id == 0 ? 92160
                                : 58880))  // 92160 (4608x20) is titan rtx,
                                           // 58880 (2944x20) is rtx 2080
                break;
        }
        lv = li;
    }

    int *d_cscColPtrTR;
    int *d_cscRowIdxTR;
    VALUE_TYPE *d_cscValTR;
    hipMalloc((void **)&d_cscColPtrTR, (n + 1) * sizeof(int));
    hipMalloc((void **)&d_cscRowIdxTR, nnzTR * sizeof(int));
    hipMalloc((void **)&d_cscValTR, nnzTR * sizeof(VALUE_TYPE));

    hipMemcpy(d_cscColPtrTR, cscColPtrTR, sizeof(int) * (n + 1),
               hipMemcpyHostToDevice);
    hipMemcpy(d_cscRowIdxTR, cscRowIdxTR, sizeof(int) * nnzTR,
               hipMemcpyHostToDevice);
    hipMemcpy(d_cscValTR, cscValTR, sizeof(VALUE_TYPE) * nnzTR,
               hipMemcpyHostToDevice);

    VALUE_TYPE *d_x;
    VALUE_TYPE *d_b;
    hipMalloc((void **)&d_x, m * sizeof(VALUE_TYPE));
    hipMalloc((void **)&d_b, m * sizeof(VALUE_TYPE));

    hipMemcpy(d_x, x, sizeof(VALUE_TYPE) * m, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(VALUE_TYPE) * m, hipMemcpyHostToDevice);

    double cal_time = 0;
    double preprocess_time = 0;
    recblocking_solver_cuda(d_cscColPtrTR, d_cscRowIdxTR, d_cscValTR, m, n,
                            nnzTR, d_x, d_b, substitution, lv, &cal_time,
                            &preprocess_time);
    hipMemcpy(x, d_x, sizeof(VALUE_TYPE) * m, hipMemcpyDeviceToHost);

    printf("Preprocess time = %.3lf ms\n", preprocess_time);
    printf("computation usetime = %.3lf ms\n", cal_time);
    printf("Performance = %.3lf gflops\n", (2 * nnzTR) / (cal_time * 1e6));

    hipFree(d_cscColPtrTR);
    hipFree(d_cscRowIdxTR);
    hipFree(d_cscValTR);
    hipFree(d_b);
    hipFree(d_x);
    free(cscColPtrTR);
    free(cscRowIdxTR);
    free(cscValTR);

    return cal_time;
}

void RunBenchmarkLowerWithCusparse(int Dof, int stencil_type, int stencil_width,
                                   int M, int N, int P, int lv) {
    constexpr int Dim = 3;

    std::vector<std::array<cusp_int, Dim>> stencil_points;
    if (stencil_type == 0) {
        for (int d = Dim - 1; d >= 0; d--) {
            for (int j = stencil_width; j > 0; j--) {
                std::array<cusp_int, Dim> pt = {0, 0, 0};
                pt[d] = -j;
                stencil_points.push_back(pt);
            }
        }
        stencil_points.push_back(std::array<cusp_int, Dim>{0, 0, 0});
    } else if (stencil_type == 1) {
        NestedLoop(
            constant_array<cusp_int, Dim>(-stencil_width),
            constant_array<cusp_int, Dim>(2 * stencil_width + 1), [&](auto pt) {
                cusp_int cnt = CartToFlat(
                    pt + stencil_width,
                    constant_array<cusp_int, Dim>(2 * stencil_width + 1));
                if (cnt < (myPow(2 * stencil_width + 1, Dim) / 2)) {
                    stencil_points.push_back(pt);
                }
            });
        stencil_points.push_back(std::array<cusp_int, Dim>{0, 0, 0});
    } else if (stencil_type == 2) {
        stencil_points.push_back(std::array<cusp_int, Dim>{0, 0, -1});
        stencil_points.push_back(std::array<cusp_int, Dim>{1, 0, -1});
        stencil_points.push_back(std::array<cusp_int, Dim>{0, 1, -1});
        stencil_points.push_back(std::array<cusp_int, Dim>{0, -1, 0});
        stencil_points.push_back(std::array<cusp_int, Dim>{1, -1, 0});
        stencil_points.push_back(std::array<cusp_int, Dim>{-1, 0, 0});
        stencil_points.push_back(std::array<cusp_int, Dim>{0, 0, 0});
    } else {
        stencil_points.push_back(std::array<cusp_int, Dim>{0, 0, -2});
        stencil_points.push_back(std::array<cusp_int, Dim>{0, -1, -1});
        stencil_points.push_back(std::array<cusp_int, Dim>{-1, 0, -1});
        stencil_points.push_back(std::array<cusp_int, Dim>{0, -2, 0});
        stencil_points.push_back(std::array<cusp_int, Dim>{0, 0, -1});
        stencil_points.push_back(std::array<cusp_int, Dim>{-1, -1, 0});
        stencil_points.push_back(std::array<cusp_int, Dim>{1, 0, -1});
        stencil_points.push_back(std::array<cusp_int, Dim>{0, -1, 0});
        stencil_points.push_back(std::array<cusp_int, Dim>{-2, 0, 0});
        stencil_points.push_back(std::array<cusp_int, Dim>{0, 1, -1});
        stencil_points.push_back(std::array<cusp_int, Dim>{1, -1, 0});
        stencil_points.push_back(std::array<cusp_int, Dim>{-1, 0, 0});
        stencil_points.push_back(std::array<cusp_int, Dim>{0, 0, 0});
    }

    // Host problem definition
    cusp_int A_num_rows = M * N * P * Dof;
    cusp_int A_nnz = 0;
    std::vector<cusp_int> hA_csrOffsets;
    std::vector<cusp_int> hA_columns;
    std::vector<double> hA_values;
    std::vector<double> hX;
    std::vector<double> hY;
    std::vector<double> hY_result;
    // 注意这里求解的是A* Y = X, 所以这里的Y是输出, X是输入

    // set A & hX
    NestedLoop(
        std::array<cusp_int, Dim>{}, std::array<cusp_int, Dim>{M, N, P},
        [&](auto loc) {
            for (int d = 0; d < Dof; d++) {
                hA_csrOffsets.push_back(A_nnz);
                cusp_int cnt = 0;
                for (auto pt : stencil_points) {
                    if (in_range(loc + pt, std::array<cusp_int, Dim>{},
                                 std::array<cusp_int, Dim>{M, N, P} - 1)) {
                        for (int k = 0; k < Dof; k++) {
                            if (pt != std::array<cusp_int, Dim>{0, 0, 0} ||
                                k == d) {
                                hA_columns.push_back(
                                    CartToFlat(
                                        loc + pt,
                                        std::array<cusp_int, Dim>{M, N, P}) *
                                        Dof +
                                    k);
                                hA_values.push_back(1.);
                                A_nnz++;
                                cnt++;
                            }
                        }
                    }
                }
                hX.push_back(cnt);
            }
        });
    hA_csrOffsets.push_back(A_nnz);

    std::cout << "A_nnz = " << A_nnz << "\n";

    // set hY
    hY.resize(A_num_rows);
    hY_result.resize(A_num_rows);
    for (cusp_int i = 0; i < A_num_rows; i++) hY_result[i] = 1.0;

    //--------------------------------------------------------------------------
    /* !!!!!! start computing SpTRSV !!!!!!!! */

    // warm up
    double solve_time = recblock_sptrsv_csr(
        A_num_rows, A_num_rows, A_nnz, hA_csrOffsets.data(), hA_columns.data(),
        hA_values.data(), hX.data(), hY.data(), lv);
    // test
    /////

    long readBytes = (sizeof(cusp_int) + sizeof(double)) * A_nnz +
                     sizeof(cusp_int) * A_num_rows +
                     sizeof(double) * A_num_rows;
    long writeBytes = sizeof(double) * A_num_rows;

    benchmark_record_map_lower[Dof - 1] = {solve_time * 1e-3 * 10,
                                           2L * A_nnz * 10,
                                           (readBytes + writeBytes) * 10, 10};

    //--------------------------------------------------------------------------
    // device result check

    int correct = 1;
    for (cusp_int i = 0; i < A_num_rows; i++) {
        if (hY[i] !=
            hY_result[i]) {  // direct doubleing point comparison is not
            correct = 0;     // reliable
            // break;
            std::cout << "i = " << i << ", hY[i] = " << hY[i]
                      << ", hY_result[i] = " << hY_result[i] << std::endl;
        }
    }
    if (correct)
        printf("recblock test PASSED\n");
    else
        printf("recblock test FAILED: wrong result\n");
    //--------------------------------------------------------------------------
    // device memory deallocation
    // log::FunctionEnd(0, 0, 0);
}

int main(int argc, char **argv) {
    std::string problems[] = {"stencilstar", "stencilbox", "stencilstarfill1",
                              "stencildiamond"};

    assert(argc > 6);
    int i = atoi(argv[1]);
    int stencil_width_0 = atoi(argv[2]);
    int dof = atoi(argv[3]) - 1;
    cusp_int M = atoi(argv[4]);
    cusp_int N = atoi(argv[5]);
    cusp_int P = atoi(argv[6]);
    int lv = argc > 7 ? atoi(argv[7]) : -1;

    int stencil_width = stencil_width_0 + 1;
    std::string problem = problems[i];

    std::cout << problem << ", width=" << stencil_width << ", dof=" << dof + 1
              << std::endl;

    std::cout << "\tmesh size=" << M << 'x' << N << 'x' << P << std::endl;
    RunBenchmarkLowerWithCusparse(dof + 1, i, stencil_width, M, N, P, lv);
    std::cout << "\t\tLower:";
    double total_time = benchmark_record_map_lower[dof].total_time;
    double total_flops_time =
        static_cast<double>(benchmark_record_map_lower[dof].flops) / total_time;
    double total_bytes_time =
        static_cast<double>(benchmark_record_map_lower[dof].bytes) / total_time;

    std::cout << dof + 1 << "," << total_time << "," << total_flops_time * 1e-9
              << "," << total_bytes_time * 1e-9 << std::endl;

    return 0;
}
